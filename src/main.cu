#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include "signal_utils.h"

#define BLOCK_SIZE 256

// CUDA kernel for denoising using moving average filter
__global__ void denoiseSignal(const float* inputSignal, float* outputSignal, int length) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx > 0 && idx < length - 1) {
        // Simple moving average filter: averaging over three consecutive points
        outputSignal[idx] = (inputSignal[idx - 1] + inputSignal[idx] + inputSignal[idx + 1]) / 3.0f;
    }
}

void processSignal(const std::string& inputPath, const std::string& denoiseOutputPath, const std::string& recoveryOutputPath) {
    std::vector<float> signal = loadSignal(inputPath);
    int length = signal.size();

    // Allocate memory for device input and output signals
    float* d_input;
    float* d_output;

    hipMalloc(&d_input, length * sizeof(float));
    hipMalloc(&d_output, length * sizeof(float));

    // Copy the signal to the device
    hipMemcpy(d_input, signal.data(), length * sizeof(float), hipMemcpyHostToDevice);

    // Set up grid and block size for CUDA kernel
    int gridSize = (length + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Run the denoising kernel
    denoiseSignal<<<gridSize, BLOCK_SIZE>>>(d_input, d_output, length);
    hipDeviceSynchronize();

    // Copy the result back to host
    std::vector<float> denoisedSignal(length);
    hipMemcpy(denoisedSignal.data(), d_output, length * sizeof(float), hipMemcpyDeviceToHost);

    // Save the denoised signal
    saveSignal(denoisedSignal, denoiseOutputPath);

    // Recover the signal by interpolation (simple linear interpolation)
    std::vector<float> recoveredSignal = recoverSignal(denoisedSignal);

    // Save the recovered signal
    saveSignal(recoveredSignal, recoveryOutputPath);

    // Clean up
    hipFree(d_input);
    hipFree(d_output);
}

int main() {
    processSignal("data/noisy_signal.csv", "output/denoised_signal.csv", "output/recovered_signal.csv");
    std::cout << "Processing completed.\n";
    return 0;
}
